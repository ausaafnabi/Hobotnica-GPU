#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <R.h>
using namespace std;


__global__ void Rkendall_gpu_atomic(const double* col1, const double* col2, const int n, const int m, unsigned long long* R){
  int row1 = blockIdx.y * blockDim.y + threadIdx.y;
  int row2 = blockIdx.x * blockDim.x + threadIdx.x;
  if (row1 < row2 && row2 < n){
    if ((col1[row1] - col1[row2]) * (col2[row1] - col2[row2]) < 0){
      atomicAdd(R, 1);
    }
  }
}


extern "C" void matrix_Kendall_distance(double* a, double* c, int* n, int* m){
  size_t dataset_column_size = *n * sizeof(double);
  size_t reverse_max_size = sizeof(unsigned long long);
  for (int col1 = 0; col1 < *m; col1++){
    double* first_column_device_ptr;
    hipMalloc(&first_column_device_ptr, dataset_column_size);
    hipMemcpy(first_column_device_ptr, a + col1 * *n, dataset_column_size, hipMemcpyHostToDevice);
    for (int col2 = col1 + 1; col2 < *m; col2 ++){
      double* second_column_device_ptr;
      hipMalloc(&second_column_device_ptr, dataset_column_size);
      hipMemcpy(second_column_device_ptr, a + col2 * *n, dataset_column_size, hipMemcpyHostToDevice);
      unsigned long long host_R = 0;
      unsigned long long* device_R;
      hipMalloc(&device_R, reverse_max_size);
      hipMemcpy(device_R, &host_R, reverse_max_size, hipMemcpyHostToDevice);
      int threads = 16;
      int blocks_in_row = (*n + threads - 1) / threads;
      int blocks_in_col = (*n + threads - 1) / threads;

      dim3 THREADS(threads, threads);
      dim3 BLOCKS(blocks_in_row, blocks_in_col);

      Rkendall_gpu_atomic<<<BLOCKS, THREADS>>>(first_column_device_ptr, second_column_device_ptr, *n, *m, device_R);
      hipDeviceSynchronize();

      hipMemcpy(&host_R, device_R, reverse_max_size, hipMemcpyDeviceToHost);
      c[col1 * *m + col2] = host_R * 2.0 / *n / (*n - 1);
      c[col2 * *m + col1] = c[col1 * *m + col2];

      hipFree(second_column_device_ptr);
      hipFree(device_R);
    }
    hipFree(first_column_device_ptr);
  }
}


extern "C" void file_Kendall_distance(double* a, int* n, int* m, char** fout){
  ofstream RESULTFILE(*fout, ios::binary|ios::app);
  size_t dataset_column_size = *n * sizeof(double);
  size_t reverse_max_size = sizeof(unsigned long long);
  for (int col1 = 0; col1 < *m; col1++){
    double* first_column_device_ptr;
    hipMalloc(&first_column_device_ptr, dataset_column_size);
    hipMemcpy(first_column_device_ptr, a + col1 * *n, dataset_column_size, hipMemcpyHostToDevice);
    for (int col2 = col1 + 1; col2 < *m; col2 ++){
      double* second_column_device_ptr;
      hipMalloc(&second_column_device_ptr, dataset_column_size);
      hipMemcpy(second_column_device_ptr, a + col2 * *n, dataset_column_size, hipMemcpyHostToDevice);
      unsigned long long host_R = 0;
      unsigned long long* device_R;
      hipMalloc(&device_R, reverse_max_size);
      hipMemcpy(device_R, &host_R, reverse_max_size, hipMemcpyHostToDevice);
      int threads = 16;
      int blocks_in_row = (*n + threads - 1) / threads;
      int blocks_in_col = (*n + threads - 1) / threads;

      dim3 THREADS(threads, threads);
      dim3 BLOCKS(blocks_in_row, blocks_in_col);

      Rkendall_gpu_atomic<<<BLOCKS, THREADS>>>(first_column_device_ptr, second_column_device_ptr, *n, *m, device_R);
      hipDeviceSynchronize();

      hipMemcpy(&host_R, device_R, reverse_max_size, hipMemcpyDeviceToHost);

      double distance = host_R * 2.0 / *n / (*n - 1);
      RESULTFILE.write((char*)&distance, sizeof(distance));

      hipFree(second_column_device_ptr);
      hipFree(device_R);
    }
    hipFree(first_column_device_ptr);
  }
  RESULTFILE.close();
}
